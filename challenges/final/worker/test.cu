#include "hip/hip_runtime.h"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""


using namespace std;
using namespace cv;

/// Global Variables





/// Function headers

//void BWFilter(Mat image);
void JustRed(unsigned char* Input_Image, int Width, int Height, int Channels, string filterType);

__global__ void red(unsigned char* image);
__global__ void BWFilter(unsigned char* Image, int Channels);
__global__ void SepiaFilter(unsigned char* Image, int Channels);
__global__ void AvatarFilter(unsigned char* Image, int Channels);

/**
 * function main
 */

int main(int argc, char** argv)
{
    Mat src; Mat dst;
    /// Load the source image
    src = imread(argv[1]);
    string name = argv[1];
    string filterType = argv[2];
    
    dst = src.clone();

    
    JustRed(dst.data, src.cols, src.rows, src.channels() , filterType);
    
    string output_name = "../public/results/"+name;
    imwrite(output_name,dst);
    
    return 0;
}

/*
void BWFilter(Mat image){

    for (int i = 0; i < image.cols; i++){
        for (int j = 0; j < image.rows; j++){
            Vec3b colors = image.at<cv::Vec3b>(j,i);
            int blue = colors[0];
            int green = colors[1];
            int red = colors[2];
            colors[0] = colors[1] = colors[2] = (red + green + blue) /3;
            image.at<cv::Vec3b>(j,i) = colors;
        }
    }


    imshow(window_name,image);
    waitKey(0);
    return;

}
*/
void JustRed(unsigned char* Input_Image, int Width, int Height, int Channels, string filterType){

    
    unsigned char* Dev_Input_Image = NULL;
   
     //allocate the memory in gpu
     hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);
    
     //copy data from CPU to GPU
     hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);
 
     dim3 Grid_Image(Width, Height);
    
     if(filterType == "bw"){
        BWFilter<<<Grid_Image, 8 >>>(Dev_Input_Image, Channels);
     }
     else if(filterType == "sepia"){
        SepiaFilter<<<Grid_Image, 8 >>>(Dev_Input_Image, Channels);
     }
     else if(filterType == "avatar"){
        AvatarFilter<<<Grid_Image, 8 >>>(Dev_Input_Image, Channels);
     }
     
     //copy processed data back to cpu from gpu
     hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);
     
     //free gpu mempry
     hipFree(Dev_Input_Image);
    
    return;

}


__global__ void SepiaFilter(unsigned char* Image, int Channels) {
    
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * Channels;

    if(Channels == 3){
        int B = Image[idx + 0];
        int G = Image[idx + 1];
        int R = Image[idx + 2];
    
        float tr = 0.393*R + 0.769*G + 0.189*B;
        float tg = 0.349*R + 0.686*G + 0.168*B;
        float tb = 0.272*R + 0.534*G + 0.131*B;

        if(tr > 255){
            tr = 255;
        }
        if(tg > 255){
            tg = 255;
        }
        if(tb > 255){
            tb = 255;
        }
        
        Image[idx + 0] = tb;
        Image[idx + 1] = tg;
        Image[idx + 2] = tr;

    }

}

__global__ void BWFilter(unsigned char* Image, int Channels) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * Channels;

    int suma = 0;
    for (int i = 0; i < Channels; i++) {
        
        suma = suma + Image[idx + i];
       
    }

    suma = suma / Channels;

    for (int i = 0; i < Channels; i++) {
        
        Image[idx + i] = suma;
       
    }
    
}


__global__ void AvatarFilter(unsigned char* Image, int Channels) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * Channels;

    if(Channels == 3){
        int B = Image[idx + 0];
        int G = Image[idx + 1];
        int R = Image[idx + 2];

        Image[idx + 0] = R;
        Image[idx + 1] = G;
        Image[idx + 2] = B;

    }
    
}


